#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <stdio.h>
#include <time.h>

#include "browAug.h"


#define TPB 64


#define N_POINTS_PER_EV 700

__device__ unsigned int generate_new_xs(float * new_xs,float * noise,const unsigned long ev_start);
__device__ void interp_linear(float * x_out,float * y_in,float * y_out, const unsigned int length_x);
__global__ void brow_Kernel(float * data_in, float * data_out,float * noise,float *ev_len_out,unsigned int nEvents);


__global__ void copyKernel(float * data_in, float * data_out,unsigned int len)
{
    const int evIdx = blockIdx.x*TPB+ threadIdx.x;
    if(evIdx<len)
    {
        for(unsigned int i = 0; i < len; i++)
            data_out[i]=data_in[i];
    }
}

__device__ unsigned int generate_new_xs(float * new_xs,float * noise,const unsigned long ev_start)
{ 
    unsigned int array_index=0,final_len;
    float curr_x_index=0;
    float x_to_interpol[N_POINTS_PER_EV]; //Obtains the new sampling points for the output (xs where it will sample) 

    for(unsigned int new_idx=0; new_idx < N_POINTS_PER_EV; new_idx++)
    {
        x_to_interpol[new_idx]=curr_x_index;
        curr_x_index += (1+noise[ev_start+new_idx]);
        if (curr_x_index < 0) //We dont allow negative indexes
            curr_x_index=0;
        if (curr_x_index > N_POINTS_PER_EV) //If the next index goes beyond the maximum, it finishes 
            break;
    }
    //array_index--;
    final_len=new_idx;
    return final_len; //new length of the event
}

__device__ void interp_linear(float * x_out,float * y_in,float * y_out, const unsigned int length_x)
{
    float p_y_i,n_y_i;
    unsigned int p_x_i,n_x_i;
    for(unsigned int i=0;i<N_POINTS_PER_EV;i++) //Every point in y_out.
    {
        if (i < length_x)
        {
            p_x_i=int(x_out[i]);n_x_i=p_x_i+1; //We find the integers that are previous and next to our xi, we will interpolated with each.
            p_y_i=y_in[p_x_i];n_y_i=y_in[n_x_i]; //Then we have the values of y_in evaluated at those integers
            y_out[i] = p_y_i + ((n_y_i-p_y_i)/(n_x_i-p_x_i)) * (x_out[i] - p_x_i); //Linear interpolation between them.

        }
        else
            y_out[i]=std::nan; //When the output event is shorter than 700, we fill with nan.
    }
}

__global__ void brow_Kernel(float * data_in, float * data_out,float * noise,float *ev_len_out,unsigned int nEvents)
{
    const int evIdx = blockIdx.x*TPB+ threadIdx.x;
    const unsigned long ev_start=ev_Idx*N_POINTS_PER_EV;
    float shared_ev_data[N_POINTS_PER_EV];float new_xs[N_POINTS_PER_EV];
    unsigned int new_length;
    if (evIdx <nEvents) //checking boundaries
    {
        for(unsigned int i=0;i<N_POINTS_PER_EV;i++) //Retrieve event to local array
            shared_ev_data[i]=data_in[ev_start+i];
        new_length=generate_new_xs(new_xs,noise,ev_start);
        ev_len_out[evIdx]=(float)new_length;
        interp_linear(new_xs,shared_ev_data,&(data_out[ev_start]),new_length);
    }
}

void BrowLauncher(hipStream_t& stream,float * data_in, float * data_out,float * noise,float *ev_len_out,unsigned int nEvents) {
  brow_Kernel<<<nEvents/TPB + 1, TPB,0,stream>>>(data_in,data_out,noise,ev_len_out,nEvents);
  hipDeviceSynchronize();
}
